#include "hip/hip_runtime.h"
﻿#define PI 3.14159265359
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

const int nx = 1025, ny = 1025, nt = 100, ns = 3, nf = 3;
const int mx = nf * nx, my = nf * ny;

const double reynolds = 200., mach = 0.2, prandtl = 0.7;
const double rhoInf = 1., cInf = 1., cylinderD = 1., heatCapacityP = 1., gamma = 1.4;

const double heatCapacityV = heatCapacityP / gamma;
const double uInf = mach * cInf;
const double dynViscosity = rhoInf * uInf * cylinderD / reynolds;
// thermal conductivity
const double lambda = dynViscosity * heatCapacityP / prandtl;
const double tempInf = cInf * cInf / (heatCapacityP * (gamma - 1));
const double eta = 0.1 / 2;

const double xLength = 4. * cylinderD;
const double yLength = 4. * cylinderD;
const double deltaX = xLength / nx;
const double deltaY = yLength / ny;
const double CFL = 0.25;
const double deltaT = CFL * deltaX;

// Derivative stencil constants
const double d_consts[] = { 1 / (2 * deltaX) , 1 / (2 * deltaY) };
__constant__ double deriv_consts[2];

void InitialiseArrays(double*, double*, double*, double*, double*, double*, double*, double*, double*, double*, double*);
void HandleError(hipError_t);
void AllocateGpuMemory(double* [], double** [], const int);

__global__ void Derix(const double*, double*);
__global__ void Deriy(const double*, double*);

int main()
{
    const int numOfVariables = 11;
    double* cylinderMask = new double[nx * ny]();
    double* uVelocity = new double[nx * ny];
    double* vVelocity = new double[nx * ny];
    double* temp = new double[nx * ny];
    double* energy = new double[nx * ny];
    double* rho = new double[nx * ny];
    double* pressure = new double[nx * ny];
    double* rou = new double[nx * ny];
    double* rov = new double[nx * ny];
    double* roe = new double[nx * ny];
    double* scp = new double[nx * ny];

    double* gpu_cylinderMask, * gpu_uVelocity, * gpu_vVelocity, * gpu_temp, * gpu_energy, * gpu_rho, * gpu_pressure, * gpu_rou, * gpu_rov, * gpu_roe, * gpu_scp;

    double* hostVariables[numOfVariables] = { cylinderMask, uVelocity, vVelocity, temp, energy, rho, pressure, rou, rov, roe, scp };
    double** gpuVariables[numOfVariables] = { &gpu_cylinderMask, &gpu_uVelocity, &gpu_vVelocity, &gpu_temp, &gpu_energy, &gpu_rho, &gpu_pressure, &gpu_rou, &gpu_rov, &gpu_roe, &gpu_scp };

    InitialiseArrays(cylinderMask, uVelocity, vVelocity, temp, energy, rho, pressure, rou, rov, roe, scp);
    AllocateGpuMemory(hostVariables, gpuVariables, numOfVariables);

    //Derix << < dim3(1, ny), 256 >> > (gpu_uVelocity, gpu_temp);
    //Deriy << < dim3(nx, 1), 256 >> > (gpu_uVelocity, gpu_temp);

    HandleError(hipMemcpy(temp, gpu_temp, sizeof(temp), hipMemcpyDeviceToHost));


    // Free memory on both host and device
    for (int i = 0; i < numOfVariables; i++) {
        HandleError(hipFree(*gpuVariables[i]) );
        delete[] hostVariables[i];
    }

    return 0;
}

void InitialiseArrays(double* cylinderMask, double* uVelocity, double* vVelocity, double* temp,
    double* energy, double* rho, double* pressure, double* rou, double* rov, double* roe, double* scp)
{

    int idx;
    double dx, dy, radSquared = cylinderD * cylinderD / 4;
    double pressInf = rhoInf * tempInf * heatCapacityP * (gamma - 1) / gamma;

    for (int j = 0; j < nx; j++) {
        for (int i = 0; i < ny; i++) {
            idx = j * ny + i;

            // Masks covers points inside circle
            dx = i * deltaX - xLength / 2;
            dy = j * deltaY - yLength / 2;
            if (dx * dx + dy * dy < radSquared) {
                cylinderMask[idx] = 1.;
            }

            uVelocity[idx] = uInf;
            // Add small velocity perturbation
            vVelocity[idx] = 0.01 * (sin(4 * PI * i * deltaX / xLength)
                + sin(7 * PI * i * deltaX / xLength))
                * exp(-(j * deltaY - yLength / 2) * (j * deltaY - yLength / 2));
            temp[idx] = tempInf;
            pressure[idx] = pressInf;
            energy[idx] = heatCapacityV * tempInf
                + 0.5 * (uInf * uInf + vVelocity[i * nx + j] * vVelocity[i * nx + j]);
            rho[idx] = rhoInf;
            rou[idx] = rhoInf * uInf;
            rov[idx] = rhoInf * vVelocity[idx];
            roe[idx] = rhoInf * energy[idx];
            scp[idx] = 1.;
        }
    }
}

void AllocateGpuMemory(double* hostVariableList[], double** gpuVariableList[], const int length) {

    // Allocate gpu memory and copy data from host arrays
    int bytes = nx * ny * sizeof(double);
    for (int i = 0; i < length; i++) {
        HandleError(hipMalloc((void**)gpuVariableList[i], bytes));
        HandleError(hipMemcpy(*gpuVariableList[i], hostVariableList[i], bytes, hipMemcpyHostToDevice));
    }

    // Allocate constant gpu memory
    HandleError(hipMemcpyToSymbol(HIP_SYMBOL(deriv_consts), d_consts, 2 * sizeof(double)));
}

void HandleError(hipError_t error) {

    if (error != hipSuccess) {
        printf("An error occured: %i: %s", error, hipGetErrorString(error));
        printf("\nExiting...");
        exit(EXIT_FAILURE);
    }
}

__global__ void Derix(const double* f, double* deriv_f) {

    __shared__ double row_f[nx + 2];

    int thrdsPerBlock = blockDim.x;
    int global_tid, shrd_mem_idx;

    // Copy row of f into shared memory
    for (int i = threadIdx.x; i < nx; i += thrdsPerBlock) {
        global_tid = ny * blockIdx.y + i;
        shrd_mem_idx = i + 1;
        row_f[shrd_mem_idx] = f[global_tid];
    }

    __syncthreads();

    // Apply periodic boundary conditions
    if (threadIdx.x == 0) {
        row_f[0] = row_f[nx];
        row_f[nx + 1] = row_f[1];
    }

    __syncthreads();

    // Calculate derivative using finite difference stencil
    for (int i = threadIdx.x; i < nx; i += thrdsPerBlock) {
        global_tid = ny * blockIdx.y + i;
        shrd_mem_idx = i + 1;
        deriv_f[global_tid] = deriv_consts[0] * (row_f[shrd_mem_idx + 1] - row_f[shrd_mem_idx - 1]);
    }
}

__global__ void Deriy(const double* f, double* deriv_f) {

    __shared__ double col_f[ny + 2];

    int thrdsPerBlock = blockDim.x;
    int global_tid, shrd_mem_idx;

    // Copy column of f into shared memory
    for (int i = threadIdx.x; i < ny; i += thrdsPerBlock) {
        global_tid = ny * i + blockIdx.x;
        shrd_mem_idx = i + 1;
        col_f[shrd_mem_idx] = f[global_tid];
    }

    __syncthreads();

    // Apply periodic boundary conditions
    if (threadIdx.x == 0) {
        col_f[0] = col_f[ny];
        col_f[ny + 1] = col_f[1];
    }

    __syncthreads();

    // Calculate derivative using finite difference stencil
    for (int i = threadIdx.x; i < ny; i += thrdsPerBlock) {
        global_tid = ny * i + blockIdx.x;
        shrd_mem_idx = i + 1;
        deriv_f[global_tid] = deriv_consts[1] * (col_f[shrd_mem_idx + 1] - col_f[shrd_mem_idx - 1]);
    }
}
